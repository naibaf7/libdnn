#include "hip/hip_runtime.h"
/* =========================================================================
   Copyright (c) 2010-2016, Institute for Microelectronics,
                            Institute for Analysis and Scientific Computing,
                            TU Wien.
   Portions of this software are copyright by UChicago Argonne, LLC.

                            -----------------
                  ViennaCL - The Vienna Computing Library
                            -----------------

   Project Head:    Karl Rupp                   rupp@iue.tuwien.ac.at

   (A list of authors and contributors can be found in the PDF manual)

   License:         MIT (X11), see file LICENSE in the base directory
============================================================================= */



/** \file tests/src/sparse.cpp  Tests sparse matrix operations.
*   \test  Tests sparse matrix operations.
**/

//
// *** System
//
#include <iostream>
#include <vector>
#include <map>
#include <cmath>

//
// *** ViennaCL
//
#include "viennacl/scalar.hpp"
#include "viennacl/compressed_matrix.hpp"
#include "viennacl/compressed_compressed_matrix.hpp"
#include "viennacl/coordinate_matrix.hpp"
#include "viennacl/ell_matrix.hpp"
#include "viennacl/sliced_ell_matrix.hpp"
#include "viennacl/hyb_matrix.hpp"
#include "viennacl/vector.hpp"
#include "viennacl/vector_proxy.hpp"
#include "viennacl/linalg/prod.hpp"
#include "viennacl/linalg/norm_2.hpp"
#include "viennacl/linalg/ilu.hpp"
#include "viennacl/linalg/detail/ilu/common.hpp"
#include "viennacl/io/matrix_market.hpp"
#include "viennacl/tools/random.hpp"



//
// -------------------------------------------------------------
//
template<typename ScalarType>
ScalarType diff(ScalarType & s1, viennacl::scalar<ScalarType> & s2)
{
   if (s1 != s2)
      return (s1 - s2) / std::max(fabs(s1), std::fabs(s2));
   return 0;
}

template<typename ScalarType>
ScalarType diff(std::vector<ScalarType> & v1, viennacl::vector<ScalarType> & v2)
{
   std::vector<ScalarType> v2_cpu(v2.size());
   viennacl::backend::finish();
   viennacl::copy(v2.begin(), v2.end(), v2_cpu.begin());

   for (unsigned int i=0;i<v1.size(); ++i)
   {
      if ( std::max( std::fabs(v2_cpu[i]), std::fabs(v1[i]) ) > 0 )
      {
        //if (std::max( std::fabs(v2_cpu[i]), std::fabs(v1[i]) ) < 1e-10 )  //absolute tolerance (avoid round-off issues)
        //  v2_cpu[i] = 0;
        //else
          v2_cpu[i] = std::fabs(v2_cpu[i] - v1[i]) / std::max( std::fabs(v2_cpu[i]), std::fabs(v1[i]) );
      }
      else
         v2_cpu[i] = 0.0;

      if (v2_cpu[i] > 0.0001)
      {
        //std::cout << "Neighbor: "      << i-1 << ": " << v1[i-1] << " vs. " << v2_cpu[i-1] << std::endl;
        std::cout << "Error at entry " << i   << ": Should: " << v1[i]   << " vs. Is: " << v2[i]   << std::endl;
        //std::cout << "Neighbor: "      << i+1 << ": " << v1[i+1] << " vs. " << v2_cpu[i+1] << std::endl;
        exit(EXIT_FAILURE);
      }
   }

   ScalarType norm_inf = 0;
   for (std::size_t i=0; i<v2_cpu.size(); ++i)
     norm_inf = std::max<ScalarType>(norm_inf, std::fabs(v2_cpu[i]));

   return norm_inf;
}


template<typename IndexT, typename NumericT, typename SparseMatrixT>
NumericT diff(std::vector<std::map<IndexT, NumericT> > & cpu_A, SparseMatrixT & vcl_A)
{
  typedef typename std::map<IndexT, NumericT>::const_iterator  RowIterator;

  std::vector<std::map<IndexT, NumericT> > from_gpu(vcl_A.size1());

  viennacl::backend::finish();
  viennacl::copy(vcl_A, from_gpu);

  NumericT error = 0;

  //step 1: compare all entries from cpu_A with vcl_A:
  for (std::size_t i=0; i<cpu_A.size(); ++i)
  {
    //std::cout << "Row " << row_it.index1() << ": " << std::endl;
    for (RowIterator it = cpu_A[i].begin(); it != cpu_A[i].end(); ++it)
    {
      //std::cout << "(" << col_it.index2() << ", " << *col_it << std::endl;
      NumericT current_error = 0;
      NumericT val_cpu_A = it->second;
      NumericT val_gpu_A = from_gpu[i][it->first];

      NumericT max_val = std::max(std::fabs(val_cpu_A), std::fabs(val_gpu_A));
      if (max_val > 0)
        current_error = std::fabs(val_cpu_A - val_gpu_A) / max_val;
      if (current_error > error)
        error = current_error;
    }
  }

  //step 2: compare all entries from gpu_matrix with cpu_matrix (sparsity pattern might differ):
  //std::cout << "ViennaCL matrix: " << std::endl;
  for (std::size_t i=0; i<from_gpu.size(); ++i)
  {
    //std::cout << "Row " << row_it.index1() << ": " << std::endl;
    for (RowIterator it = from_gpu[i].begin(); it != from_gpu[i].end(); ++it)
    {
      //std::cout << "(" << col_it.index2() << ", " << *col_it << std::endl;
      NumericT current_error = 0;
      NumericT val_gpu_A = it->second;
      NumericT val_cpu_A = cpu_A[i][it->first];

      NumericT max_val = std::max(std::fabs(val_cpu_A), std::fabs(val_gpu_A));
      if (max_val > 0)
        current_error = std::fabs(val_cpu_A - val_gpu_A) / max_val;
      if (current_error > error)
        error = current_error;
    }
  }

  return error;
}


template<typename NumericT, typename VCL_MatrixT, typename Epsilon, typename STLVectorT, typename VCLVectorT>
int strided_matrix_vector_product_test(Epsilon epsilon,
                                       STLVectorT & result,     STLVectorT const & rhs,
                                       VCLVectorT & vcl_result, VCLVectorT & vcl_rhs)
{
  typedef typename std::map<unsigned int, NumericT>::const_iterator    RowIterator;
    int retval = EXIT_SUCCESS;

    std::vector<std::map<unsigned int, NumericT> > std_A(5);
    std_A[0][0] = NumericT(2.0); std_A[0][2] = NumericT(-1.0);
    std_A[1][0] = NumericT(3.0); std_A[1][2] = NumericT(-5.0);
    std_A[2][1] = NumericT(5.0); std_A[2][2] = NumericT(-2.0);
    std_A[3][2] = NumericT(1.0); std_A[3][3] = NumericT(-6.0);
    std_A[4][1] = NumericT(7.0); std_A[4][2] = NumericT(-5.0);
    //the following computes project(result, slice(1, 3, 5)) = prod(std_A, project(rhs, slice(3, 2, 4)));
    for (std::size_t i=0; i<5; ++i)
    {
      NumericT val = 0;
      for (RowIterator it = std_A[i].begin(); it != std_A[i].end(); ++it)
        val += it->second * rhs[3 + 2*it->first];
      result[1 + 3*i] = val;
    }

    VCL_MatrixT vcl_sparse_matrix2;
    viennacl::copy(std_A, vcl_sparse_matrix2);
    viennacl::vector<NumericT> vec(4);
    vec(0) = rhs[3];
    vec(1) = rhs[5];
    vec(2) = rhs[7];
    vec(3) = rhs[9];
    viennacl::project(vcl_result, viennacl::slice(1, 3, 5)) = viennacl::linalg::prod(vcl_sparse_matrix2, viennacl::project(vcl_rhs, viennacl::slice(3, 2, 4)));

    if ( std::fabs(diff(result, vcl_result)) > epsilon )
    {
      std::cout << "# Error at operation: matrix-vector product with strided vectors, part 1" << std::endl;
      std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
      retval = EXIT_FAILURE;
    }
    vcl_result(1)  = NumericT(1.0);
    vcl_result(4)  = NumericT(1.0);
    vcl_result(7)  = NumericT(1.0);
    vcl_result(10) = NumericT(1.0);
    vcl_result(13) = NumericT(1.0);

    viennacl::project(vcl_result, viennacl::slice(1, 3, 5)) = viennacl::linalg::prod(vcl_sparse_matrix2, vec);

    if ( std::fabs(diff(result, vcl_result)) > epsilon )
    {
      std::cout << "# Error at operation: matrix-vector product with strided vectors, part 2" << std::endl;
      std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
      retval = EXIT_FAILURE;
    }

    return retval;
}


template< typename NumericT, typename VCL_MATRIX, typename Epsilon >
int resize_test(Epsilon const& epsilon)
{
   int retval = EXIT_SUCCESS;

   std::vector<std::map<unsigned int, NumericT> > std_A(5);
   VCL_MATRIX vcl_matrix;

   std_A[0][0] = NumericT(10.0); std_A[0][1] = NumericT(0.1); std_A[0][2] = NumericT(0.2); std_A[0][3] = NumericT(0.3); std_A[0][4] = NumericT(0.4);
   std_A[1][0] = NumericT(1.0);  std_A[1][1] = NumericT(1.1); std_A[1][2] = NumericT(1.2); std_A[1][3] = NumericT(1.3); std_A[1][4] = NumericT(1.4);
   std_A[2][0] = NumericT(2.0);  std_A[2][1] = NumericT(2.1); std_A[2][2] = NumericT(2.2); std_A[2][3] = NumericT(2.3); std_A[2][4] = NumericT(2.4);
   std_A[3][0] = NumericT(3.0);  std_A[3][1] = NumericT(3.1); std_A[3][2] = NumericT(3.2); std_A[3][3] = NumericT(3.3); std_A[3][4] = NumericT(3.4);
   std_A[4][0] = NumericT(4.0);  std_A[4][1] = NumericT(4.1); std_A[4][2] = NumericT(4.2); std_A[4][3] = NumericT(4.3); std_A[4][4] = NumericT(4.4);

   viennacl::copy(std_A, vcl_matrix);
   std::vector<std::map<unsigned int, NumericT> > std_B(std_A.size());
   viennacl::copy(vcl_matrix, std_B);

   std::cout << "Checking for equality after copy..." << std::endl;
    if ( std::fabs(diff(std_A, vcl_matrix)) > epsilon )
    {
        std::cout << "# Error at operation: equality after copy with sparse matrix" << std::endl;
        std::cout << "  diff: " << std::fabs(diff(std_A, vcl_matrix)) << std::endl;
        return EXIT_FAILURE;
    }

   std::cout << "Testing resize to larger..." << std::endl;
   std_A.resize(10);
   std_A[0][0] = NumericT(10.0); std_A[0][1] = NumericT(0.1); std_A[0][2] = NumericT(0.2); std_A[0][3] = NumericT(0.3); std_A[0][4] = NumericT(0.4);
   std_A[1][0] = NumericT( 1.0); std_A[1][1] = NumericT(1.1); std_A[1][2] = NumericT(1.2); std_A[1][3] = NumericT(1.3); std_A[1][4] = NumericT(1.4);
   std_A[2][0] = NumericT( 2.0); std_A[2][1] = NumericT(2.1); std_A[2][2] = NumericT(2.2); std_A[2][3] = NumericT(2.3); std_A[2][4] = NumericT(2.4);
   std_A[3][0] = NumericT( 3.0); std_A[3][1] = NumericT(3.1); std_A[3][2] = NumericT(3.2); std_A[3][3] = NumericT(3.3); std_A[3][4] = NumericT(3.4);
   std_A[4][0] = NumericT( 4.0); std_A[4][1] = NumericT(4.1); std_A[4][2] = NumericT(4.2); std_A[4][3] = NumericT(4.3); std_A[4][4] = NumericT(4.4);

   vcl_matrix.resize(10, 10, true);

    if ( std::fabs(diff(std_A, vcl_matrix)) > epsilon )
    {
        std::cout << "# Error at operation: resize (to larger) with sparse matrix" << std::endl;
        std::cout << "  diff: " << std::fabs(diff(std_A, vcl_matrix)) << std::endl;
        return EXIT_FAILURE;
    }

   std_A[5][5] = NumericT(5.5); std_A[5][6] = NumericT(5.6); std_A[5][7] = NumericT(5.7); std_A[5][8] = NumericT(5.8); std_A[5][9] = NumericT(5.9);
   std_A[6][5] = NumericT(6.5); std_A[6][6] = NumericT(6.6); std_A[6][7] = NumericT(6.7); std_A[6][8] = NumericT(6.8); std_A[6][9] = NumericT(6.9);
   std_A[7][5] = NumericT(7.5); std_A[7][6] = NumericT(7.6); std_A[7][7] = NumericT(7.7); std_A[7][8] = NumericT(7.8); std_A[7][9] = NumericT(7.9);
   std_A[8][5] = NumericT(8.5); std_A[8][6] = NumericT(8.6); std_A[8][7] = NumericT(8.7); std_A[8][8] = NumericT(8.8); std_A[8][9] = NumericT(8.9);
   std_A[9][5] = NumericT(9.5); std_A[9][6] = NumericT(9.6); std_A[9][7] = NumericT(9.7); std_A[9][8] = NumericT(9.8); std_A[9][9] = NumericT(9.9);
   viennacl::copy(std_A, vcl_matrix);

   std::cout << "Testing resize to smaller..." << std::endl;
   std_A.clear();
   std_A.resize(7);
   std_A[0][0] = NumericT(10.0); std_A[0][1] = NumericT(0.1); std_A[0][2] = NumericT(0.2); std_A[0][3] = NumericT(0.3); std_A[0][4] = NumericT(0.4);
   std_A[1][0] = NumericT( 1.0); std_A[1][1] = NumericT(1.1); std_A[1][2] = NumericT(1.2); std_A[1][3] = NumericT(1.3); std_A[1][4] = NumericT(1.4);
   std_A[2][0] = NumericT( 2.0); std_A[2][1] = NumericT(2.1); std_A[2][2] = NumericT(2.2); std_A[2][3] = NumericT(2.3); std_A[2][4] = NumericT(2.4);
   std_A[3][0] = NumericT( 3.0); std_A[3][1] = NumericT(3.1); std_A[3][2] = NumericT(3.2); std_A[3][3] = NumericT(3.3); std_A[3][4] = NumericT(3.4);
   std_A[4][0] = NumericT( 4.0); std_A[4][1] = NumericT(4.1); std_A[4][2] = NumericT(4.2); std_A[4][3] = NumericT(4.3); std_A[4][4] = NumericT(4.4);
   std_A[5][5] = NumericT( 5.5); std_A[5][6] = NumericT(5.6); //std_A[5][7] = NumericT(5.7); std_A[5][8] = NumericT(5.8); std_A[5][9] = NumericT(5.9);
   std_A[6][5] = NumericT( 6.5); std_A[6][6] = NumericT(6.6); //std_A[6][7] = NumericT(6.7); std_A[6][8] = NumericT(6.8); std_A[6][9] = NumericT(6.9);

   vcl_matrix.resize(7, 7);

   //std::cout << std_A << std::endl;
    if ( std::fabs(diff(std_A, vcl_matrix)) > epsilon )
    {
        std::cout << "# Error at operation: resize (to smaller) with sparse matrix" << std::endl;
        std::cout << "  diff: " << std::fabs(diff(std_A, vcl_matrix)) << std::endl;
        retval = EXIT_FAILURE;
    }

   std::vector<NumericT> std_vec(std_A.size(), NumericT(3.1415));
   viennacl::vector<NumericT> vcl_vec(std_A.size());


  std::cout << "Testing unit lower triangular solve: compressed_matrix" << std::endl;
  viennacl::copy(std_vec, vcl_vec);

  std::cout << "STL..." << std::endl;
  //boost::numeric::ublas::inplace_solve((ublas_matrix), ublas_vec, boost::numeric::ublas::unit_lower_tag());
  for (std::size_t i=1; i<std_A.size(); ++i)
    for (typename std::map<unsigned int, NumericT>::const_iterator it = std_A[i].begin(); it != std_A[i].end(); ++it)
    {
      if (it->first < static_cast<unsigned int>(i))
        std_vec[i] -= it->second * std_vec[it->first];
      else
        continue;
    }

  std::cout << "ViennaCL..." << std::endl;
  viennacl::linalg::inplace_solve((vcl_matrix), vcl_vec, viennacl::linalg::unit_lower_tag());

  if ( std::fabs(diff(std_vec, vcl_vec)) > epsilon )
  {
      std::cout << "# Error at operation: unit lower triangular solve" << std::endl;
      std::cout << "  diff: " << std::fabs(diff(std_vec, vcl_vec)) << std::endl;
      retval = EXIT_FAILURE;
  }
  return retval;
}


//
// -------------------------------------------------------------
//
template< typename NumericT, typename Epsilon >
int test(Epsilon const& epsilon)
{
  viennacl::tools::uniform_random_numbers<NumericT> randomNumber;

  std::cout << "Testing resizing of compressed_matrix..." << std::endl;
  int retval = resize_test<NumericT, viennacl::compressed_matrix<NumericT> >(epsilon);
  if (retval != EXIT_SUCCESS)
    return retval;

  // --------------------------------------------------------------------------
  std::vector<NumericT> rhs;
  std::vector<NumericT> result;
  std::vector<std::map<unsigned int, NumericT> > std_matrix;

  if (viennacl::io::read_matrix_market_file(std_matrix, "../examples/testdata/mat65k.mtx") == EXIT_FAILURE)
  {
    std::cout << "Error reading Matrix file" << std::endl;
    return EXIT_FAILURE;
  }

  //unsigned int cg_mat_size = cg_mat.size();
  std::cout << "done reading matrix" << std::endl;


  rhs.resize(std_matrix.size());
  for (std::size_t i=0; i<rhs.size(); ++i)
  {
    std_matrix[i][static_cast<unsigned int>(i)] = NumericT(0.5);   // Get rid of round-off errors by making row-sums unequal to zero:
    rhs[i] = NumericT(1) + randomNumber();
  }

  // add some random numbers to the double-compressed matrix:
  std::vector<std::map<unsigned int, NumericT> > std_cc_matrix(std_matrix.size());
  std_cc_matrix[42][199] = NumericT(3.1415);
  std_cc_matrix[31][69] = NumericT(2.71);
  std_cc_matrix[23][32] = NumericT(6);
  std_cc_matrix[177][57] = NumericT(4);
  std_cc_matrix[21][97] = NumericT(-4);
  std_cc_matrix[92][25] = NumericT(2);
  std_cc_matrix[89][62] = NumericT(11);
  std_cc_matrix[ 1][ 7] = NumericT(8);
  std_cc_matrix[85][41] = NumericT(13);
  std_cc_matrix[66][28] = NumericT(8);
  std_cc_matrix[21][74] = NumericT(-2);
  viennacl::tools::sparse_matrix_adapter<NumericT> adapted_std_cc_matrix(std_cc_matrix, std_matrix.size(), std_matrix.size());


  result = rhs;


  viennacl::vector<NumericT> vcl_rhs(rhs.size());
  viennacl::vector<NumericT> vcl_result(result.size());
  viennacl::vector<NumericT> vcl_result2(result.size());
  viennacl::compressed_matrix<NumericT> vcl_compressed_matrix(rhs.size(), rhs.size());
  viennacl::compressed_compressed_matrix<NumericT> vcl_compressed_compressed_matrix(rhs.size(), rhs.size());
  viennacl::coordinate_matrix<NumericT> vcl_coordinate_matrix(rhs.size(), rhs.size());
  viennacl::ell_matrix<NumericT> vcl_ell_matrix;
  viennacl::sliced_ell_matrix<NumericT> vcl_sliced_ell_matrix;
  viennacl::hyb_matrix<NumericT> vcl_hyb_matrix;

  viennacl::copy(rhs.begin(), rhs.end(), vcl_rhs.begin());
  viennacl::copy(std_matrix, vcl_compressed_matrix);
  viennacl::copy(adapted_std_cc_matrix, vcl_compressed_compressed_matrix);
  viennacl::copy(std_matrix, vcl_coordinate_matrix);

  // --------------------------------------------------------------------------
  std::cout << "Testing products: STL" << std::endl;
  result = viennacl::linalg::prod(std_matrix, rhs);

  std::cout << "Testing products: compressed_matrix" << std::endl;
  vcl_result = viennacl::linalg::prod(vcl_compressed_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with compressed_matrix" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  std::cout << "Testing products: compressed_matrix, strided vectors" << std::endl;
  retval = strided_matrix_vector_product_test<NumericT, viennacl::compressed_matrix<NumericT> >(epsilon, result, rhs, vcl_result, vcl_rhs);
  if (retval != EXIT_SUCCESS)
    return retval;

  result = rhs;
  result = viennacl::linalg::prod(std_matrix, rhs);
  for (std::size_t i=0; i<result.size(); ++i) result[i] += rhs[i];
  vcl_result = vcl_rhs;
  vcl_result += viennacl::linalg::prod(vcl_compressed_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with compressed_matrix (+=)" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  result = rhs;
  result = viennacl::linalg::prod(std_matrix, rhs);
  for (std::size_t i=0; i<result.size(); ++i) result[i] = rhs[i] - result[i];
  vcl_result = vcl_rhs;
  vcl_result -= viennacl::linalg::prod(vcl_compressed_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with compressed_matrix (-=)" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  //
  // Triangular solvers for A \ b:
  //

  std::cout << "Testing unit upper triangular solve: compressed_matrix" << std::endl;
  result = rhs;
  viennacl::copy(result, vcl_result);
  //boost::numeric::ublas::inplace_solve(trans(ublas_matrix_trans), result, boost::numeric::ublas::unit_upper_tag());
  for (std::size_t i2=0; i2<std_matrix.size(); ++i2)
  {
    std::size_t row = std_matrix.size() - i2 - 1;
    for (typename std::map<unsigned int, NumericT>::const_iterator it = std_matrix[row].begin(); it != std_matrix[row].end(); ++it)
    {
      if (it->first > static_cast<unsigned int>(row))
        result[row] -= it->second * result[it->first];
      else
        continue;
    }
  }

  viennacl::linalg::inplace_solve(vcl_compressed_matrix, vcl_result, viennacl::linalg::unit_upper_tag());

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: unit upper triangular solve with compressed_matrix" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  ////////////////////////////

  std::cout << "Testing upper triangular solve: compressed_matrix" << std::endl;
  result = rhs;
  viennacl::copy(result, vcl_result);
  //boost::numeric::ublas::inplace_solve(trans(ublas_matrix_trans), result, boost::numeric::ublas::upper_tag());
  for (std::size_t i2=0; i2<std_matrix.size(); ++i2)
  {
    std::size_t row = std_matrix.size() - i2 - 1;
    NumericT diag = 0;
    for (typename std::map<unsigned int, NumericT>::const_iterator it = std_matrix[row].begin(); it != std_matrix[row].end(); ++it)
    {
      if (it->first > static_cast<unsigned int>(row))
        result[row] -= it->second * result[it->first];
      else if (it->first == static_cast<unsigned int>(row))
        diag = it->second;
      else
        continue;
    }
    result[row] /= diag;
  }

  viennacl::linalg::inplace_solve(vcl_compressed_matrix, vcl_result, viennacl::linalg::upper_tag());

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: upper triangular solve with compressed_matrix" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  ////////////////////////////

  std::cout << "Testing unit lower triangular solve: compressed_matrix" << std::endl;
  result = rhs;
  viennacl::copy(result, vcl_result);
  //boost::numeric::ublas::inplace_solve(trans(ublas_matrix_trans), result, boost::numeric::ublas::unit_lower_tag());
  for (std::size_t i=1; i<std_matrix.size(); ++i)
    for (typename std::map<unsigned int, NumericT>::const_iterator it = std_matrix[i].begin(); it != std_matrix[i].end(); ++it)
    {
      if (it->first < static_cast<unsigned int>(i))
        result[i] -= it->second * result[it->first];
      else
        continue;
    }
  viennacl::linalg::inplace_solve(vcl_compressed_matrix, vcl_result, viennacl::linalg::unit_lower_tag());


  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: unit lower triangular solve with compressed_matrix" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }


  std::cout << "Testing lower triangular solve: compressed_matrix" << std::endl;
  result = rhs;
  viennacl::copy(result, vcl_result);
  //boost::numeric::ublas::inplace_solve(trans(ublas_matrix_trans), result, boost::numeric::ublas::lower_tag());
  for (std::size_t i=0; i<std_matrix.size(); ++i)
  {
    NumericT diag = 0;
    for (typename std::map<unsigned int, NumericT>::const_iterator it = std_matrix[i].begin(); it != std_matrix[i].end(); ++it)
    {
      if (it->first < static_cast<unsigned int>(i))
        result[i] -= it->second * result[it->first];
      else if (it->first == static_cast<unsigned int>(i))
        diag = it->second;
      else
        continue;
    }
    result[i] /= diag;
  }
  viennacl::linalg::inplace_solve(vcl_compressed_matrix, vcl_result, viennacl::linalg::lower_tag());


  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: lower triangular solve with compressed_matrix" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }



  //
  // Triangular solvers for A^T \ b
  //
  std::vector<std::map<unsigned int, NumericT> > std_matrix_trans(std_matrix.size());

  // compute transpose:
  for (std::size_t i=0; i<std_matrix.size(); ++i)
    for (typename std::map<unsigned int, NumericT>::const_iterator it  = std_matrix[i].begin(); it != std_matrix[i].end(); ++it)
      std_matrix_trans[i][it->first] = it->second;

  std::cout << "Testing transposed unit upper triangular solve: compressed_matrix" << std::endl;
  result = rhs;
  viennacl::copy(result, vcl_result);
  //boost::numeric::ublas::inplace_solve(trans(ublas_matrix), result, boost::numeric::ublas::unit_upper_tag());
  for (std::size_t i2=0; i2<std_matrix_trans.size(); ++i2)
  {
    std::size_t row = std_matrix_trans.size() - i2 - 1;
    for (typename std::map<unsigned int, NumericT>::const_iterator it = std_matrix_trans[row].begin(); it != std_matrix_trans[row].end(); ++it)
    {
      if (it->first > static_cast<unsigned int>(row))
        result[row] -= it->second * result[it->first];
      else
        continue;
    }
  }
  viennacl::linalg::inplace_solve(trans(vcl_compressed_matrix), vcl_result, viennacl::linalg::unit_upper_tag());

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: unit upper triangular solve with compressed_matrix" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  /////////////////////////

  std::cout << "Testing transposed upper triangular solve: compressed_matrix" << std::endl;
  result = rhs;
  viennacl::copy(result, vcl_result);
  //boost::numeric::ublas::inplace_solve(trans(ublas_matrix), result, boost::numeric::ublas::upper_tag());
  for (std::size_t i2=0; i2<std_matrix_trans.size(); ++i2)
  {
    std::size_t row = std_matrix_trans.size() - i2 - 1;
    NumericT diag = 0;
    for (typename std::map<unsigned int, NumericT>::const_iterator it = std_matrix_trans[row].begin(); it != std_matrix_trans[row].end(); ++it)
    {
      if (it->first > static_cast<unsigned int>(row))
        result[row] -= it->second * result[it->first];
      else if (it->first == static_cast<unsigned int>(row))
        diag = it->second;
      else
        continue;
    }
    result[row] /= diag;
  }
  viennacl::linalg::inplace_solve(trans(vcl_compressed_matrix), vcl_result, viennacl::linalg::upper_tag());

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: upper triangular solve with compressed_matrix" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  /////////////////////////

  std::cout << "Testing transposed unit lower triangular solve: compressed_matrix" << std::endl;
  result = rhs;
  viennacl::copy(result, vcl_result);
  //boost::numeric::ublas::inplace_solve(trans(ublas_matrix), result, boost::numeric::ublas::unit_lower_tag());
  for (std::size_t i=1; i<std_matrix_trans.size(); ++i)
    for (typename std::map<unsigned int, NumericT>::const_iterator it = std_matrix_trans[i].begin(); it != std_matrix_trans[i].end(); ++it)
    {
      if (it->first < static_cast<unsigned int>(i))
        result[i] -= it->second * result[it->first];
      else
        continue;
    }
  viennacl::linalg::inplace_solve(trans(vcl_compressed_matrix), vcl_result, viennacl::linalg::unit_lower_tag());

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: unit lower triangular solve with compressed_matrix" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  /////////////////////////

  std::cout << "Testing transposed lower triangular solve: compressed_matrix" << std::endl;
  result = rhs;
  viennacl::copy(result, vcl_result);
  //boost::numeric::ublas::inplace_solve(trans(ublas_matrix), result, boost::numeric::ublas::lower_tag());
  for (std::size_t i=0; i<std_matrix_trans.size(); ++i)
  {
    NumericT diag = 0;
    for (typename std::map<unsigned int, NumericT>::const_iterator it = std_matrix_trans[i].begin(); it != std_matrix_trans[i].end(); ++it)
    {
      if (it->first < static_cast<unsigned int>(i))
        result[i] -= it->second * result[it->first];
      else if (it->first == static_cast<unsigned int>(i))
        diag = it->second;
      else
        continue;
    }
    result[i] /= diag;
  }
  viennacl::linalg::inplace_solve(trans(vcl_compressed_matrix), vcl_result, viennacl::linalg::lower_tag());

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: lower triangular solve with compressed_matrix" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }


  //
  /////////////////////////
  //


  std::cout << "Testing products: compressed_compressed_matrix" << std::endl;
  result     = viennacl::linalg::prod(std_cc_matrix, rhs);
  vcl_result = viennacl::linalg::prod(vcl_compressed_compressed_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with compressed_compressed_matrix (=)" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  {
    std::vector<std::map<unsigned int, NumericT> > temp(vcl_compressed_compressed_matrix.size1());
    viennacl::copy(vcl_compressed_compressed_matrix, temp);

    // check that entries are correct by computing the product again:
    result     = viennacl::linalg::prod(temp, rhs);

    if ( std::fabs(diff(result, vcl_result)) > epsilon )
    {
      std::cout << "# Error at operation: matrix-vector product with compressed_compressed_matrix (after copy back)" << std::endl;
      std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
      return EXIT_FAILURE;
    }

  }

  result = rhs;
  result = viennacl::linalg::prod(std_cc_matrix, rhs);
  for (std::size_t i=0; i<result.size(); ++i) result[i] += rhs[i];
  vcl_result = vcl_rhs;
  vcl_result += viennacl::linalg::prod(vcl_compressed_compressed_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with compressed_compressed_matrix (+=)" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  result = rhs;
  result = viennacl::linalg::prod(std_cc_matrix, rhs);
  for (std::size_t i=0; i<result.size(); ++i) result[i] = rhs[i] - result[i];
  vcl_result = vcl_rhs;
  vcl_result -= viennacl::linalg::prod(vcl_compressed_compressed_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with compressed_compressed_matrix (-=)" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }


  //
  /////////////////////////
  //


  std::cout << "Testing products: coordinate_matrix" << std::endl;
  result     = viennacl::linalg::prod(std_matrix, rhs);
  vcl_result = viennacl::linalg::prod(vcl_coordinate_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with coordinate_matrix" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  std::cout << "Testing products: coordinate_matrix, strided vectors" << std::endl;
  //std::cout << " --> SKIPPING <--" << std::endl;
  retval = strided_matrix_vector_product_test<NumericT, viennacl::coordinate_matrix<NumericT> >(epsilon, result, rhs, vcl_result, vcl_rhs);
  if (retval != EXIT_SUCCESS)
    return retval;

  result = rhs;
  result = viennacl::linalg::prod(std_matrix, rhs);
  for (std::size_t i=0; i<result.size(); ++i) result[i] += rhs[i];
  vcl_result = vcl_rhs;
  vcl_result += viennacl::linalg::prod(vcl_coordinate_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with coordinate_matrix (+=)" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  result = rhs;
  result = viennacl::linalg::prod(std_matrix, rhs);
  for (std::size_t i=0; i<result.size(); ++i) result[i] = rhs[i] - result[i];
  vcl_result = vcl_rhs;
  vcl_result -= viennacl::linalg::prod(vcl_coordinate_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with coordinate_matrix (-=)" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  //
  /////////////////////////
  //


  //std::cout << "Copying ell_matrix" << std::endl;
  viennacl::copy(std_matrix, vcl_ell_matrix);
  std_matrix.clear();
  viennacl::copy(vcl_ell_matrix, std_matrix);// just to check that it works


  std::cout << "Testing products: ell_matrix" << std::endl;
  result     = viennacl::linalg::prod(std_matrix, rhs);
  vcl_result.clear();
  vcl_result = viennacl::linalg::prod(vcl_ell_matrix, vcl_rhs);
  //viennacl::linalg::prod_impl(vcl_ell_matrix, vcl_rhs, vcl_result);
  //std::cout << vcl_result << "\n";
  //std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
  //std::cout << "First entry of result vector: " << vcl_result[0] << std::endl;

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with ell_matrix" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  std::cout << "Testing products: ell_matrix, strided vectors" << std::endl;
  retval = strided_matrix_vector_product_test<NumericT, viennacl::ell_matrix<NumericT> >(epsilon, result, rhs, vcl_result, vcl_rhs);
  if (retval != EXIT_SUCCESS)
    return retval;

  result = rhs;
  result = viennacl::linalg::prod(std_matrix, rhs);
  for (std::size_t i=0; i<result.size(); ++i) result[i] += rhs[i];
  vcl_result = vcl_rhs;
  vcl_result += viennacl::linalg::prod(vcl_ell_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with ell_matrix (+=)" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  result = rhs;
  result = viennacl::linalg::prod(std_matrix, rhs);
  for (std::size_t i=0; i<result.size(); ++i) result[i] = rhs[i] - result[i];
  vcl_result = vcl_rhs;
  vcl_result -= viennacl::linalg::prod(vcl_ell_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with ell_matrix (-=)" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  //
  /////////////////////////
  //


  //std::cout << "Copying sliced_ell_matrix" << std::endl;
  viennacl::copy(std_matrix, vcl_sliced_ell_matrix);

  std::cout << "Testing products: sliced_ell_matrix" << std::endl;
  result     = viennacl::linalg::prod(std_matrix, rhs);
  vcl_result.clear();
  vcl_result = viennacl::linalg::prod(vcl_sliced_ell_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with sliced_ell_matrix" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  std::cout << "Testing products: sliced_ell_matrix, strided vectors" << std::endl;
  retval = strided_matrix_vector_product_test<NumericT, viennacl::sliced_ell_matrix<NumericT> >(epsilon, result, rhs, vcl_result, vcl_rhs);
  if (retval != EXIT_SUCCESS)
    return retval;

  result = rhs;
  result = viennacl::linalg::prod(std_matrix, rhs);
  for (std::size_t i=0; i<result.size(); ++i) result[i] += rhs[i];
  vcl_result = vcl_rhs;
  vcl_result += viennacl::linalg::prod(vcl_sliced_ell_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with sliced_ell_matrix (+=)" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  result = rhs;
  result = viennacl::linalg::prod(std_matrix, rhs);
  for (std::size_t i=0; i<result.size(); ++i) result[i] = rhs[i] - result[i];
  vcl_result = vcl_rhs;
  vcl_result -= viennacl::linalg::prod(vcl_sliced_ell_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with sliced_ell_matrix (-=)" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  //
  /////////////////////////
  //


  //std::cout << "Copying hyb_matrix" << std::endl;
  viennacl::copy(std_matrix, vcl_hyb_matrix);
  std_matrix.clear();
  viennacl::copy(vcl_hyb_matrix, std_matrix);// just to check that it works
  viennacl::copy(std_matrix, vcl_hyb_matrix);

  std::cout << "Testing products: hyb_matrix" << std::endl;
  result     = viennacl::linalg::prod(std_matrix, rhs);
  vcl_result.clear();
  vcl_result = viennacl::linalg::prod(vcl_hyb_matrix, vcl_rhs);
  //viennacl::linalg::prod_impl(vcl_hyb_matrix, vcl_rhs, vcl_result);
  //std::cout << vcl_result << "\n";
  //std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
  //std::cout << "First entry of result vector: " << vcl_result[0] << std::endl;

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with hyb_matrix" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  std::cout << "Testing products: hyb_matrix, strided vectors" << std::endl;
  retval = strided_matrix_vector_product_test<NumericT, viennacl::hyb_matrix<NumericT> >(epsilon, result, rhs, vcl_result, vcl_rhs);
  if (retval != EXIT_SUCCESS)
    return retval;

  result = rhs;
  result = viennacl::linalg::prod(std_matrix, rhs);
  for (std::size_t i=0; i<result.size(); ++i) result[i] += rhs[i];
  vcl_result = vcl_rhs;
  vcl_result += viennacl::linalg::prod(vcl_hyb_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with hyb_matrix (+=)" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  result = rhs;
  result = viennacl::linalg::prod(std_matrix, rhs);
  for (std::size_t i=0; i<result.size(); ++i) result[i] = rhs[i] - result[i];
  vcl_result = vcl_rhs;
  vcl_result -= viennacl::linalg::prod(vcl_hyb_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with hyb_matrix (-=)" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }


  // --------------------------------------------------------------------------
  // --------------------------------------------------------------------------
  NumericT alpha = static_cast<NumericT>(2.786);
  NumericT beta = static_cast<NumericT>(1.432);
  copy(rhs.begin(), rhs.end(), vcl_rhs.begin());
  copy(result.begin(), result.end(), vcl_result.begin());
  copy(result.begin(), result.end(), vcl_result2.begin());

  std::cout << "Testing scaled additions of products and vectors" << std::endl;
  std::vector<NumericT> result2(result);
  result2 = viennacl::linalg::prod(std_matrix, rhs);
  for (std::size_t i=0; i<result.size(); ++i)
    result[i] = alpha * result2[i] + beta * result[i];
  vcl_result2 = alpha * viennacl::linalg::prod(vcl_compressed_matrix, vcl_rhs) + beta * vcl_result;

  if ( std::fabs(diff(result, vcl_result2)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product (compressed_matrix) with scaled additions" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result2)) << std::endl;
    return EXIT_FAILURE;
  }


  vcl_result2.clear();
  vcl_result2 = alpha * viennacl::linalg::prod(vcl_coordinate_matrix, vcl_rhs) + beta * vcl_result;

  if ( std::fabs(diff(result, vcl_result2)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product (coordinate_matrix) with scaled additions" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result2)) << std::endl;
    return EXIT_FAILURE;
  }

  vcl_result2.clear();
  vcl_result2 = alpha * viennacl::linalg::prod(vcl_ell_matrix, vcl_rhs) + beta * vcl_result;

  if ( std::fabs(diff(result, vcl_result2)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product (ell_matrix) with scaled additions" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result2)) << std::endl;
    return EXIT_FAILURE;
  }

  vcl_result2.clear();
  vcl_result2 = alpha * viennacl::linalg::prod(vcl_hyb_matrix, vcl_rhs) + beta * vcl_result;

  if ( std::fabs(diff(result, vcl_result2)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product (hyb_matrix) with scaled additions" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result2)) << std::endl;
    return EXIT_FAILURE;
  }

  ////////////// Test of .clear() ////////////////
  std_matrix.clear();

  std::cout << "Testing products after clear(): compressed_matrix" << std::endl;
  vcl_compressed_matrix.clear();
  result     = std::vector<NumericT>(result.size(), NumericT(1));
  result     = viennacl::linalg::prod(std_matrix, rhs);
  vcl_result = viennacl::linalg::prod(vcl_compressed_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with compressed_matrix" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  std::cout << "Testing products after clear(): compressed_compressed_matrix" << std::endl;
  vcl_compressed_compressed_matrix.clear();
  result     = std::vector<NumericT>(result.size(), NumericT(1));
  result     = viennacl::linalg::prod(std_matrix, rhs);
  vcl_result = viennacl::linalg::prod(vcl_compressed_compressed_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with compressed_compressed_matrix" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  std::cout << "Testing products after clear(): coordinate_matrix" << std::endl;
  vcl_coordinate_matrix.clear();
  result     = std::vector<NumericT>(result.size(), NumericT(1));
  result     = viennacl::linalg::prod(std_matrix, rhs);
  vcl_result = viennacl::linalg::prod(vcl_coordinate_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with coordinate_matrix" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  std::cout << "Testing products after clear(): ell_matrix" << std::endl;
  vcl_ell_matrix.clear();
  result     = std::vector<NumericT>(result.size(), NumericT(1));
  result     = viennacl::linalg::prod(std_matrix, rhs);
  vcl_result = viennacl::linalg::prod(vcl_ell_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with ell_matrix" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  std::cout << "Testing products after clear(): hyb_matrix" << std::endl;
  vcl_hyb_matrix.clear();
  result     = std::vector<NumericT>(result.size(), NumericT(1));
  result     = viennacl::linalg::prod(std_matrix, rhs);
  vcl_result = viennacl::linalg::prod(vcl_hyb_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with hyb_matrix" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }

  std::cout << "Testing products after clear(): sliced_ell_matrix" << std::endl;
  vcl_sliced_ell_matrix.clear();
  result     = std::vector<NumericT>(result.size(), NumericT(1));
  result     = viennacl::linalg::prod(std_matrix, rhs);
  vcl_result = viennacl::linalg::prod(vcl_sliced_ell_matrix, vcl_rhs);

  if ( std::fabs(diff(result, vcl_result)) > epsilon )
  {
    std::cout << "# Error at operation: matrix-vector product with sliced_ell_matrix" << std::endl;
    std::cout << "  diff: " << std::fabs(diff(result, vcl_result)) << std::endl;
    return EXIT_FAILURE;
  }


  // --------------------------------------------------------------------------
  return retval;
}
//
// -------------------------------------------------------------
//
int main()
{
  std::cout << std::endl;
  std::cout << "----------------------------------------------" << std::endl;
  std::cout << "----------------------------------------------" << std::endl;
  std::cout << "## Test :: Sparse Matrices" << std::endl;
  std::cout << "----------------------------------------------" << std::endl;
  std::cout << "----------------------------------------------" << std::endl;
  std::cout << std::endl;

  int retval = EXIT_SUCCESS;

  std::cout << std::endl;
  std::cout << "----------------------------------------------" << std::endl;
  std::cout << std::endl;
  {
    typedef float NumericT;
    NumericT epsilon = static_cast<NumericT>(1E-4);
    std::cout << "# Testing setup:" << std::endl;
    std::cout << "  eps:     " << epsilon << std::endl;
    std::cout << "  numeric: float" << std::endl;
    retval = test<NumericT>(epsilon);
    if ( retval == EXIT_SUCCESS )
        std::cout << "# Test passed" << std::endl;
    else
        return retval;
  }
  std::cout << std::endl;
  std::cout << "----------------------------------------------" << std::endl;
  std::cout << std::endl;

#ifdef VIENNACL_WITH_OPENCL
  if ( viennacl::ocl::current_device().double_support() )
#endif
  {
    {
      typedef double NumericT;
      NumericT epsilon = 1.0E-12;
      std::cout << "# Testing setup:" << std::endl;
      std::cout << "  eps:     " << epsilon << std::endl;
      std::cout << "  numeric: double" << std::endl;
      retval = test<NumericT>(epsilon);
      if ( retval == EXIT_SUCCESS )
        std::cout << "# Test passed" << std::endl;
      else
        return retval;
    }
    std::cout << std::endl;
    std::cout << "----------------------------------------------" << std::endl;
    std::cout << std::endl;
  }
#ifdef VIENNACL_WITH_OPENCL
  else
    std::cout << "No double precision support, skipping test..." << std::endl;
#endif


  std::cout << std::endl;
  std::cout << "------- Test completed --------" << std::endl;
  std::cout << std::endl;

  return retval;
}
